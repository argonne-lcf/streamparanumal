#include "hip/hip_runtime.h"
/*

See LICENSE file.

*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "meshBasis.hpp"

#define READ_TO_REGISTER 1

#define CUDA_SYMBOL(a) a


__forceinline__ __device__ __host__  int ijN(const int i, const int j, const int N){

  return i + j*N;

}

__forceinline__ __device__ __host__ int ijkN(const int i, const int j, const int k, const int N){

  return i + j*N + k*N*N;

}

__forceinline__ __device__ __host__ int ijklN(const int i, const int j, const int k, const int l, const int N){

  return i + j*N + k*N*N + l*N*N*N;

}

// switch:
// 1 to use CUDA 10.0 stream recording
// 0 to use traditional enqueing of kernels
#define USE_GRAPH 0

#define MAX_DOFS_1D 16
#define MAX_QUAD_1D 16

#define MAX_HALF_DOFS_1D 8
#define MAX_HALF_QUAD_1D 8

#define HALF_DOFS_1D ((NUM_DOFS_1D+1)/2)
#define HALF_QUAD_1D ((NUM_QUAD_1D+1)/2)

#define p_padCubNq 0
// ((NUM_QUAD_1D%4) ? 0:1)

#define NUM_DOFS_2D (NUM_DOFS_1D*NUM_DOFS_1D)
#define NUM_DOFS_3D (NUM_DOFS_1D*NUM_DOFS_1D*NUM_DOFS_1D)

#define NUM_QUAD_2D (NUM_QUAD_1D*NUM_QUAD_1D)
#define NUM_QUAD_3D (NUM_QUAD_1D*NUM_QUAD_1D*NUM_QUAD_1D)

#define p_Nvgeo 1
#define p_JWID 0

void matrixPrint(int Nrows, int Ncols, dfloat *A, const char *mess){
#if 0
  printf("%s = [\n", mess);
  for(int i=0;i<Nrows;++i){
    for(int a=0;a<Ncols;++a){
      printf(" % e", A[i*Ncols+a]);
    }
    printf("\n");
  }
  printf("]\n");
#endif
}

__constant__ dfloat const_DofToQuad[MAX_QUAD_1D*MAX_DOFS_1D];
__constant__ dfloat const_oddDofToQuad[MAX_HALF_QUAD_1D*MAX_HALF_DOFS_1D];
__constant__ dfloat const_evenDofToQuad[MAX_HALF_QUAD_1D*MAX_HALF_DOFS_1D];

void randAlloc(int N, dfloat **h_a, dfloat **c_a){

  *h_a = (dfloat*) calloc(N, sizeof(dfloat));

  for(int n=0;n<N;++n)
    h_a[0][n] = drand48();

  hipMalloc(c_a, N*sizeof(dfloat));

  hipMemcpy(c_a[0], h_a[0], N*sizeof(dfloat), hipMemcpyHostToDevice);

}

__global__ void nothingKernel(){  }

__global__ void nothingVerboseKernel(int n, dfloat *creOut, dfloat *cimOut){


  if(n==-1 || n==-7098 || n==1023 || n==3521){ // this will never be true

    dfloat cre = threadIdx.x + blockIdx.x*blockDim.x;
    dfloat cim = threadIdx.y + blockIdx.x*blockDim.y;

#pragma unroll 1
    for(int i=0;i<1;++i){
      dfloat tmpre = cre*cre-cim*cim;
      dfloat tmpim = 2.*cre*cim;

      cre = tmpre;
      cim = tmpim;

    }

    creOut[0] = cre;
    
  }
}

template <int NUM_DOFS_1D, int NUM_QUAD_1D, int p_Nblock >
  __forceinline__ __device__ 
  void BK1MonolithicDevice(const int numElements,
			   const int element,
			   const dfloat * __restrict__ op,
			   const dfloat * __restrict__ DofToQuad,
			   dfloat s_Ap[p_Nblock][NUM_QUAD_1D][NUM_QUAD_1D][NUM_QUAD_1D+p_padCubNq],
			   dfloat * __restrict__ r_Ap){
  
  dfloat r_tmp[NUM_QUAD_1D];
  
  const int t   = threadIdx.x;
  const int blk = threadIdx.y;
  
  // assumes barrier before s_Ap was used last
  // TRY REARRANING THIS
  
  // transform in 'c'
  if(t<NUM_DOFS_2D){
    const int a = t%NUM_DOFS_1D;
    const int b = t/NUM_DOFS_1D;
    
#pragma unroll
    for(int k=0;k<NUM_QUAD_1D;++k){
      dfloat res = 0;
      
#pragma unroll
      for(int c=0;c<NUM_DOFS_1D;++c){
	int kc = ijN(c,k,NUM_DOFS_1D);		
	res  += DofToQuad[kc]*r_Ap[c];
      }
      
      s_Ap[blk][k][b][a]  = res;
    }
  }
  
  __syncthreads();

  // transform in 'b'
  if(t<NUM_DOFS_1D*NUM_QUAD_1D){
    const int a = t%NUM_DOFS_1D;
    const int k = t/NUM_DOFS_1D;
    
#pragma unroll
    for(int b=0;b<NUM_DOFS_1D;++b){
      r_tmp[b]  = s_Ap[blk][k][b][a];
    }
    
#pragma unroll
    for(int j=0;j<NUM_QUAD_1D;++j){
      dfloat res = 0;
      
#pragma unroll
      for(int b=0;b<NUM_DOFS_1D;++b){
	int jb = ijN(b,j,NUM_DOFS_1D);
	res  += DofToQuad[jb]*r_tmp[b];
      }
      s_Ap[blk][k][j][a] = res;
    }
  }
  
  __syncthreads();

  // transform in 'a'
  {
    const int j = t%NUM_QUAD_1D;
    const int k = t/NUM_QUAD_1D;
    
#pragma unroll
    for(int a=0;a<NUM_DOFS_1D;++a){
      r_tmp[a]  = s_Ap[blk][k][j][a];
    }
    
#pragma unroll
    for(int i=0;i<NUM_QUAD_1D;++i){
      dfloat res = 0;
      
#pragma unroll
      for(int a=0;a<NUM_DOFS_1D;++a){
	int ia = ijN(a,i,NUM_DOFS_1D);
	res  += DofToQuad[ia]*r_tmp[a];
      }
	
      int gid = ijklN(i,j,k,element, NUM_QUAD_1D);
      
      dfloat WJ = (element<numElements) ? op[gid]: 0;
      
      r_Ap[i] = WJ*res;
    }
    
#pragma unroll
    for(int a=0;a<NUM_DOFS_1D;++a){
      dfloat res = 0;
      
#pragma unroll
      for(int i=0;i<NUM_QUAD_1D;++i){
	int ia = ijN(a,i,NUM_DOFS_1D);
	res  += DofToQuad[ia]*r_Ap[i];
      }
    
      s_Ap[blk][k][j][a] = res;
    }
  }
  
  __syncthreads();

  
  // test in 'b'
  if(t<NUM_DOFS_1D*NUM_QUAD_1D){
    const int a = t%NUM_DOFS_1D;
    const int k = t/NUM_DOFS_1D;
    
    for(int j=0;j<NUM_QUAD_1D;++j){
      r_tmp[j]  = s_Ap[blk][k][j][a];
    }
    
#pragma unroll
    for(int b=0;b<NUM_DOFS_1D;++b){
      dfloat res = 0;
      
#pragma unroll
      for(int j=0;j<NUM_QUAD_1D;++j){
	int jb = ijN(b,j,NUM_DOFS_1D);
	res += DofToQuad[jb]*r_tmp[j];
      }
      
      s_Ap[blk][k][b][a] = res;
    }
  }
  
  __syncthreads();

  // test in 'c'
  if(t<NUM_DOFS_2D){
    const int a = t%NUM_DOFS_1D;
    const int b = t/NUM_DOFS_1D;
    
    for(int k=0;k<NUM_QUAD_1D;++k){
      r_tmp[k]  = s_Ap[blk][k][b][a];
    }

#pragma unroll
    for(int c=0;c<NUM_DOFS_1D;++c){
      dfloat res = 0; 
      
#pragma unroll
      for(int k=0;k<NUM_QUAD_1D;++k){
	int kc = ijN(c,k,NUM_DOFS_1D);
	res += DofToQuad[kc]*r_tmp[k];
      }

      r_Ap[c] = res;
    }
  }

#if USE_CONTIGUOUS_OUTPUT==1
  
  __syncthreads();

  // write to shared
  if(t<NUM_DOFS_2D){

#pragma unroll
    for(int c=0;c<NUM_DOFS_1D;++c){
      const int a = t%NUM_DOFS_1D;
      const int b = t/NUM_DOFS_1D;
      int id = ijklN(a,b,c,blk, NUM_DOFS_1D);
      s_Ap[0][0][0][id] = r_Ap[c];
    }
  }

#endif
  __syncthreads();
  
}

template <int NUM_DOFS_1D, int NUM_QUAD_1D, int p_Nblock >
  __forceinline__ __device__ 
  void BK1OddEvenDevice(const int numElements,
			const int element,
			const dfloat * __restrict__ op,
			const dfloat * __restrict__ oddDofToQuad,
			const dfloat * __restrict__ evenDofToQuad,
			dfloat s_Ap[p_Nblock][NUM_QUAD_1D][NUM_QUAD_1D][NUM_QUAD_1D+p_padCubNq],
			dfloat * __restrict__ r_Ap){

  dfloat r_tmpOdd[HALF_QUAD_1D];
  dfloat r_tmpEven[HALF_QUAD_1D];

  const int t   = threadIdx.x;
  const int blk = threadIdx.y;

  // assumes barrier before s_Ap was used last
  
  // transform in 'c'
  if(t<NUM_DOFS_2D){
    const int a = t%NUM_DOFS_1D;
    const int b = t/NUM_DOFS_1D;
    
#pragma unroll
    for(int c=0;c<HALF_DOFS_1D;++c){
      r_tmpOdd[c]  = r_Ap[c] + r_Ap[NUM_DOFS_1D-1-c];
      r_tmpEven[c] = r_Ap[c] - r_Ap[NUM_DOFS_1D-1-c];
    }
    
    if(NUM_DOFS_1D%2)
      r_tmpOdd[HALF_DOFS_1D-1] *= 0.5f;
    
#pragma unroll
    for(int k=0;k<HALF_QUAD_1D;++k){
      dfloat resOdd = 0, resEven = 0;
      
#pragma unroll
      for(int c=0;c<HALF_DOFS_1D;++c){
	int kc = ijN(c,k,HALF_DOFS_1D);		
	resOdd  += oddDofToQuad[kc]*r_tmpOdd[c];
	resEven += evenDofToQuad[kc]*r_tmpEven[c];
      }
      
      s_Ap[blk][NUM_QUAD_1D-1-k][b][a] = resOdd - resEven;
      s_Ap[blk][k][b][a]               = resOdd + resEven;
    }
  }
  
  __syncthreads();

  // transform in 'b'
  if(t<NUM_DOFS_1D*NUM_QUAD_1D){
    
    const int a = t%NUM_DOFS_1D;
    const int k = t/NUM_DOFS_1D;
    
#pragma unroll
    for(int b=0;b<HALF_DOFS_1D;++b){
      dfloat ApOdd  = s_Ap[blk][k][b][a];
      dfloat ApEven = s_Ap[blk][k][NUM_DOFS_1D-1-b][a];
      r_tmpOdd[b]  = ApOdd + ApEven;
      r_tmpEven[b] = ApOdd - ApEven;
    }
    
    if(NUM_DOFS_1D%2)
      r_tmpOdd[HALF_DOFS_1D-1] *= 0.5f;
    
#pragma unroll
    for(int j=0;j<HALF_QUAD_1D;++j){
      dfloat resOdd = 0, resEven = 0;
      
#pragma unroll
      for(int b=0;b<HALF_DOFS_1D;++b){
	int jb = ijN(b,j,HALF_DOFS_1D);
	resOdd  += oddDofToQuad[jb]*r_tmpOdd[b];
	resEven += evenDofToQuad[jb]*r_tmpEven[b];
      }
      
      s_Ap[blk][k][NUM_QUAD_1D-1-j][a] = resOdd-resEven;
      s_Ap[blk][k][j][a]               = resOdd+resEven;
    }
  }
  
  __syncthreads();

  // transform in 'a'
  {
    const int j = t%NUM_QUAD_1D;
    const int k = t/NUM_QUAD_1D;
    
#pragma unroll
    for(int a=0;a<HALF_DOFS_1D;++a){
      dfloat ApOdd  = s_Ap[blk][k][j][a];
      dfloat ApEven = s_Ap[blk][k][j][NUM_DOFS_1D-1-a];
      r_tmpOdd[a]  = ApOdd + ApEven;
      r_tmpEven[a] = ApOdd - ApEven;
    }
    
    if(NUM_DOFS_1D%2)
      r_tmpOdd[HALF_DOFS_1D-1] *= 0.5f;
    
#pragma unroll
    for(int i=0;i<HALF_QUAD_1D;++i){
      dfloat resOdd = 0, resEven = 0;
      
#pragma unroll
      for(int a=0;a<HALF_DOFS_1D;++a){
	int ia = ijN(a,i,HALF_DOFS_1D);
	resOdd  += oddDofToQuad[ia]*r_tmpOdd[a];
	resEven += evenDofToQuad[ia]*r_tmpEven[a];
      }
      
      int gid1 = ijklN(NUM_QUAD_1D-1-i,j,k,element, NUM_QUAD_1D);
      int gid2 = ijklN(i,j,k,element, NUM_QUAD_1D);

      dfloat WJ1 = (element<numElements) ? op[gid1]:0;
      dfloat WJ2 = (element<numElements) ? op[gid2]:0;

#if 0
      s_Ap[blk][k][j][NUM_QUAD_1D-1-i] = WJ1*(resOdd-resEven);
      s_Ap[blk][k][j][i]               = WJ2*(resOdd+resEven);
#else
      r_Ap[NUM_QUAD_1D-1-i] = WJ1*(resOdd-resEven);
      r_Ap[i]               = WJ2*(resOdd+resEven);
#endif
    }
  }
  
  __syncthreads();
  
  {
    const int j = t%NUM_QUAD_1D;
    const int k = t/NUM_QUAD_1D;
    
#pragma unroll
    for(int i=0;i<HALF_QUAD_1D;++i){
#if 0
      dfloat ApOdd  = s_Ap[blk][k][j][i];
      dfloat ApEven = s_Ap[blk][k][j][NUM_QUAD_1D-1-i];
#else
      dfloat ApOdd  = r_Ap[i];
      dfloat ApEven = r_Ap[NUM_QUAD_1D-1-i];
#endif
      
      r_tmpOdd[i]  = ApOdd + ApEven;
      r_tmpEven[i] = ApOdd - ApEven;
    }
    
    if(NUM_QUAD_1D%2)
      r_tmpOdd[HALF_QUAD_1D-1] *= 0.5f;
    
#pragma unroll
    for(int a=0;a<HALF_DOFS_1D;++a){
      dfloat resOdd = 0, resEven = 0;
      
#pragma unroll
      for(int i=0;i<HALF_QUAD_1D;++i){
	int ia = ijN(a,i,HALF_DOFS_1D);
	resOdd  += oddDofToQuad[ia]*r_tmpOdd[i];
	resEven += evenDofToQuad[ia]*r_tmpEven[i];
      }
      
      s_Ap[blk][k][j][NUM_DOFS_1D-1-a] = resOdd-resEven;
      s_Ap[blk][k][j][a]               = resOdd+resEven;
    }
  }

  __syncthreads();
  
  // test in 'b'
  if(t<NUM_DOFS_1D*NUM_QUAD_1D){
    const int a = t%NUM_DOFS_1D;
    const int k = t/NUM_DOFS_1D;
    
    for(int j=0;j<HALF_QUAD_1D;++j){
      dfloat ApOdd  = s_Ap[blk][k][j][a];
      dfloat ApEven = s_Ap[blk][k][NUM_QUAD_1D-1-j][a];
      r_tmpOdd[j]  = ApOdd + ApEven;
      r_tmpEven[j] = ApOdd - ApEven;
    }
    
    if(NUM_QUAD_1D%2)
      r_tmpOdd[HALF_QUAD_1D-1] *= 0.5f;
    
#pragma unroll
    for(int b=0;b<HALF_DOFS_1D;++b){
      dfloat resOdd = 0, resEven = 0;
      
#pragma unroll
      for(int j=0;j<HALF_QUAD_1D;++j){
	int jb = ijN(b,j,HALF_DOFS_1D);
	resOdd  += oddDofToQuad[jb]*r_tmpOdd[j];
	resEven += evenDofToQuad[jb]*r_tmpEven[j];
      }
      
      s_Ap[blk][k][NUM_DOFS_1D-1-b][a] = resOdd - resEven;
      s_Ap[blk][k][b][a]               = resOdd + resEven;
    }
  }
  
  __syncthreads();

  // test in 'c'
  if(t<NUM_DOFS_2D){
    const int a = t%NUM_DOFS_1D;
    const int b = t/NUM_DOFS_1D;
    
    for(int k=0;k<HALF_QUAD_1D;++k){
      dfloat ApOdd  = s_Ap[blk][k][b][a];
      dfloat ApEven = s_Ap[blk][NUM_QUAD_1D-1-k][b][a];
      r_tmpOdd[k]  = ApOdd + ApEven;
      r_tmpEven[k] = ApOdd - ApEven;
    }
    
    if(NUM_QUAD_1D%2)
      r_tmpOdd[HALF_QUAD_1D-1] *= 0.5f;
    
#pragma unroll
    for(int c=0;c<HALF_DOFS_1D;++c){
      dfloat resOdd = 0, resEven = 0;
      
#pragma unroll
      for(int k=0;k<HALF_QUAD_1D;++k){
	int kc = ijN(c,k,HALF_DOFS_1D);
	resOdd  += oddDofToQuad[kc]*r_tmpOdd[k];
	resEven += evenDofToQuad[kc]*r_tmpEven[k];
      }
      
      r_Ap[NUM_DOFS_1D-1-c] = resOdd - resEven;
      r_Ap[c]               = resOdd + resEven;
    }
  }

#if USE_CONTIGUOUS_OUTPUT==1
  __syncthreads();

  // write to shared
  if(t<NUM_DOFS_2D){

    const int a = t%NUM_DOFS_1D;
    const int b = t/NUM_DOFS_1D;
    
#pragma unroll
    for(int c=0;c<NUM_DOFS_1D;++c){
      int id = ijklN(a,b,c,blk, NUM_DOFS_1D);
      s_Ap[0][0][0][id] = r_Ap[c];
    }
  }
#endif
  
  __syncthreads();

}


template <int NUM_DOFS_1D, int NUM_QUAD_1D, int p_Nblock >
  __global__ void BK1RegisterKernel(const int numElements,
				    const dfloat * __restrict__ op,
				    const dfloat * __restrict__ oddDofToQuad,
				    const dfloat * __restrict__ evenDofToQuad,
				    const dfloat * __restrict__ solIn,
				    dfloat * __restrict__ solOut){
  
  __shared__ dfloat s_tmp1[p_Nblock][NUM_QUAD_1D][NUM_QUAD_1D][NUM_QUAD_1D+p_padCubNq];

  dfloat r_oddDofToQuad[HALF_QUAD_1D*HALF_DOFS_1D];
  dfloat r_evenDofToQuad[HALF_QUAD_1D*HALF_DOFS_1D];

  dfloat r_Aq[NUM_QUAD_1D];

  const unsigned int t = threadIdx.x;
  const int blk = threadIdx.y;
  
  const int element = blockIdx.x*p_Nblock + blk;
  
  const unsigned int a = t%NUM_DOFS_1D;
  const unsigned int b = t/NUM_DOFS_1D;

  
#if READ_TO_REGISTER==1
  if(element < numElements && t<NUM_DOFS_2D){
    for(int c=0;c<NUM_DOFS_1D;++c){
      int id = ijklN(a,b,c,element, NUM_DOFS_1D); 
      
      r_Aq[c] = solIn[id];
    }
  }
#else
  int n = t + blk*NUM_QUAD_2D;
  
  while(n<p_Nblock*NUM_DOFS_3D){
    
    int id = n + blockIdx.x*p_Nblock*NUM_DOFS_3D;
    if(id<numElements*NUM_DOFS_3D){
      s_tmp1[0][0][0][n] = solIn[id];
    }
    n+=NUM_QUAD_2D*p_Nblock;
  }
  
#endif
  {
    __shared__ dfloat s_oddDofToQuad[HALF_DOFS_1D*HALF_QUAD_1D];
    __shared__ dfloat s_evenDofToQuad[HALF_QUAD_1D*HALF_DOFS_1D];
    
    if(blk==0)
      for(int n=t;n<HALF_DOFS_1D*HALF_QUAD_1D;n+=NUM_QUAD_2D){
	s_oddDofToQuad[n] = oddDofToQuad[n];
	s_evenDofToQuad[n] = evenDofToQuad[n];
      }
    
    __syncthreads();
    
    // now copy shared data to thread local register arrays
    for(int n=0;n<HALF_DOFS_1D*HALF_QUAD_1D;++n){
      r_oddDofToQuad[n] = s_oddDofToQuad[n];
      r_evenDofToQuad[n] = s_evenDofToQuad[n];
    }
  }

#if READ_TO_REGISTER==0
  if(t<NUM_DOFS_2D)
    for(int c=0;c<NUM_DOFS_1D;++c)
      r_Aq[c] = s_tmp1[blk][c][b][a];

  __syncthreads();
#endif
  
  BK1OddEvenDevice <NUM_DOFS_1D, NUM_QUAD_1D, p_Nblock>
    (numElements, element, op, r_oddDofToQuad, r_evenDofToQuad, s_tmp1, r_Aq);

#if USE_CONTIGUOUS_OUTPUT==0
  if(element<numElements && t<NUM_DOFS_2D){
#pragma unroll
    for(int c=0;c<NUM_DOFS_1D;++c){
      int id = ijklN(a,b,c,element,NUM_DOFS_1D);
      solOut[id] = r_Aq[c];
    }
  }
#else
  
  int n = t + blk*NUM_QUAD_2D;
  
  while(n<p_Nblock*NUM_DOFS_3D){
    
    int id = n + blockIdx.x*p_Nblock*NUM_DOFS_3D;
    if(id<numElements*NUM_DOFS_3D){
      solOut[id] = s_tmp1[0][0][0][n];
    }
    n+=NUM_QUAD_2D*p_Nblock;
  }
#endif  
}


template <int NUM_DOFS_1D, int NUM_QUAD_1D, int p_Nblock >
  __global__ void BK1SharedKernel(const int numElements,
				  const dfloat * __restrict__ op,
				  const dfloat * __restrict__ oddDofToQuad,
				  const dfloat * __restrict__ evenDofToQuad,
				  const dfloat * __restrict__ solIn,
				  dfloat * __restrict__ solOut){
  
  __shared__ dfloat s_tmp1[p_Nblock][NUM_QUAD_1D][NUM_QUAD_1D][NUM_QUAD_1D+p_padCubNq];
  __shared__ dfloat s_oddDofToQuad[HALF_QUAD_1D*HALF_DOFS_1D];
  __shared__ dfloat s_evenDofToQuad[HALF_QUAD_1D*HALF_DOFS_1D];

  dfloat r_Aq[NUM_QUAD_1D];

  const unsigned int t = threadIdx.x;
  const int blk = threadIdx.y;
  
  const int element = blockIdx.x*p_Nblock + blk;
  
  const unsigned int a = t%NUM_DOFS_1D;
  const unsigned int b = t/NUM_DOFS_1D;

#if READ_TO_REGISTER==1
  if(element < numElements && t<NUM_DOFS_2D){
    for(int c=0;c<NUM_DOFS_1D;++c){
      
      int id = ijklN(a,b,c,element,NUM_DOFS_1D);
      
      r_Aq[c] = solIn[id];
    }
  }
#else
  int n = t + blk*NUM_QUAD_2D;
  
  while(n<p_Nblock*NUM_DOFS_3D){
    
    int id = n + blockIdx.x*p_Nblock*NUM_DOFS_3D;
    if(id<numElements*NUM_DOFS_3D){
      s_tmp1[0][0][0][n] = solIn[id];
    }
    n+=NUM_QUAD_2D*p_Nblock;
  }
#endif
  
  if(blk==0)
    for(int n=t;n<HALF_DOFS_1D*HALF_QUAD_1D;n+=NUM_QUAD_2D){
      s_oddDofToQuad[n] = oddDofToQuad[n];
      s_evenDofToQuad[n] = evenDofToQuad[n];
    }

  __syncthreads();

#if READ_TO_REGISTER==0
  if(t<NUM_DOFS_2D)
    for(int c=0;c<NUM_DOFS_1D;++c)
      r_Aq[c] = s_tmp1[blk][c][b][a];

  __syncthreads();
#endif
  
  BK1OddEvenDevice  <NUM_DOFS_1D, NUM_QUAD_1D, p_Nblock>
    (numElements, element, op, s_oddDofToQuad, s_evenDofToQuad, s_tmp1, r_Aq);

#if USE_CONTIGUOUS_OUTPUT==0
  if(element<numElements && t<NUM_DOFS_2D){
#pragma unroll
    for(int c=0;c<NUM_DOFS_1D;++c){
      int id = ijklN(a,b,c,element,NUM_DOFS_1D);
      solOut[id] = r_Aq[c];
    }
  }
#else
  
  int n = t + blk*NUM_QUAD_2D;
  
  while(n<p_Nblock*NUM_DOFS_3D){
    
    int id = n + blockIdx.x*p_Nblock*NUM_DOFS_3D;
    if(id<numElements*NUM_DOFS_3D){
      solOut[id] = s_tmp1[0][0][0][n];
    }
    n+=NUM_QUAD_2D*p_Nblock;
  }
#endif  
}

template <int NUM_DOFS_1D, int NUM_QUAD_1D, int p_Nblock >
  __global__ void BK1ConstantKernel(const int numElements,
				    const dfloat * __restrict__ op,
				    const dfloat * __restrict__ oddDofToQuad,
				    const dfloat * __restrict__ evenDofToQuad,
				    const dfloat * __restrict__ solIn,
				    dfloat * __restrict__ solOut){
  
  __shared__ dfloat s_tmp1[p_Nblock][NUM_QUAD_1D][NUM_QUAD_1D][NUM_QUAD_1D+p_padCubNq];

  dfloat r_Aq[NUM_QUAD_1D];

  const unsigned int t = threadIdx.x;
  const int blk = threadIdx.y;
  
  const int element = blockIdx.x*p_Nblock + blk;
  
  const unsigned int a = t%NUM_DOFS_1D;
  const unsigned int b = t/NUM_DOFS_1D;

#if READ_TO_REGISTER==1
  if(element < numElements && t<NUM_DOFS_2D){
    for(int c=0;c<NUM_DOFS_1D;++c){
      int id = ijklN(a,b,c,element,NUM_DOFS_1D);
      
      r_Aq[c] = solIn[id];
    }
  }
#else
  int n = t + blk*NUM_QUAD_2D;
  
  while(n<p_Nblock*NUM_DOFS_3D){
    
    int id = n + blockIdx.x*p_Nblock*NUM_DOFS_3D;
    if(id<numElements*NUM_DOFS_3D){
      s_tmp1[0][0][0][n] = solIn[id];
    }
    n+=NUM_QUAD_2D*p_Nblock;
  }
#endif

  __syncthreads();

#if READ_TO_REGISTER==0
  if(t<NUM_DOFS_2D)
    for(int c=0;c<NUM_DOFS_1D;++c)
      r_Aq[c] = s_tmp1[blk][c][b][a];

  __syncthreads();
#endif
  
  BK1OddEvenDevice  <NUM_DOFS_1D, NUM_QUAD_1D, p_Nblock>
    (numElements, element, op, const_oddDofToQuad, const_evenDofToQuad, s_tmp1, r_Aq);

#if USE_CONTIGUOUS_OUTPUT==0
  
  if(element<numElements && t<NUM_DOFS_2D){
#pragma unroll
    for(int c=0;c<NUM_DOFS_1D;++c){
      int id = ijklN(a,b,c,element,NUM_DOFS_1D);
      solOut[id] = r_Aq[c];
    }
  }

#else
  
  int n = t + blk*NUM_QUAD_2D;
  
  while(n<p_Nblock*NUM_DOFS_3D){
    
    int id = n + blockIdx.x*p_Nblock*NUM_DOFS_3D;
    if(id<numElements*NUM_DOFS_3D){
      solOut[id] = s_tmp1[0][0][0][n];
    }
    n+=NUM_QUAD_2D*p_Nblock;
  }
#endif  

  
}

template <int NUM_DOFS_1D, int NUM_QUAD_1D, int p_Nblock >
  __global__ void BK1GlobalKernel(const int numElements,
				  const dfloat * __restrict__ op,
				  const dfloat * __restrict__ oddDofToQuad,
				  const dfloat * __restrict__ evenDofToQuad,
				  const dfloat * __restrict__ solIn,
				  dfloat * __restrict__ solOut){
  
  __shared__ dfloat s_tmp1[p_Nblock][NUM_QUAD_1D][NUM_QUAD_1D][NUM_QUAD_1D+p_padCubNq];

  dfloat r_Aq[NUM_QUAD_1D];

  const unsigned int t = threadIdx.x;
  const int blk = threadIdx.y;
  
  const int element = blockIdx.x*p_Nblock + blk;
  
  const unsigned int a = t%NUM_DOFS_1D;
  const unsigned int b = t/NUM_DOFS_1D;

#if READ_TO_REGISTER==1
  if(element < numElements && t<NUM_DOFS_2D){
    for(int c=0;c<NUM_DOFS_1D;++c){
      int id = ijklN(a,b,c,element,NUM_DOFS_1D);
      
      r_Aq[c] = solIn[id];
    }
  }
#else
  int n = t + blk*NUM_QUAD_2D;
  
  while(n<p_Nblock*NUM_DOFS_3D){
    
    int id = n + blockIdx.x*p_Nblock*NUM_DOFS_3D;
    if(id<numElements*NUM_DOFS_3D){
      s_tmp1[0][0][0][n] = solIn[id];
    }
    n+=NUM_QUAD_2D*p_Nblock;
  }
#endif

  __syncthreads();

#if READ_TO_REGISTER==0
  if(t<NUM_DOFS_2D)
    for(int c=0;c<NUM_DOFS_1D;++c)
      r_Aq[c] = s_tmp1[blk][c][b][a];

  __syncthreads();
#endif
  
  BK1OddEvenDevice  <NUM_DOFS_1D, NUM_QUAD_1D, p_Nblock>
    (numElements, element, op, oddDofToQuad, evenDofToQuad, s_tmp1, r_Aq);

#if USE_CONTIGUOUS_OUTPUT==0
  
  if(element<numElements && t<NUM_DOFS_2D){
#pragma unroll
    for(int c=0;c<NUM_DOFS_1D;++c){
      int id = ijklN(a,b,c,element,NUM_DOFS_1D);
      solOut[id] = r_Aq[c];
    }
  }

#else
  
  int n = t + blk*NUM_QUAD_2D;
  
  while(n<p_Nblock*NUM_DOFS_3D){
    
    int id = n + blockIdx.x*p_Nblock*NUM_DOFS_3D;
    if(id<numElements*NUM_DOFS_3D){
      solOut[id] = s_tmp1[0][0][0][n];
    }
    n+=NUM_QUAD_2D*p_Nblock;
  }
#endif  

  
}



template <int NUM_DOFS_1D, int NUM_QUAD_1D, int p_Nblock >
  __global__ void BK1MonolithicGlobalKernel(const int numElements,
					    const dfloat * __restrict__ op,
					    const dfloat * __restrict__ DofToQuad,
					    const dfloat * __restrict__ evenDofToQuad,
					    const dfloat * __restrict__ solIn,
					    dfloat * __restrict__ solOut){
  
  __shared__ dfloat s_tmp1[p_Nblock][NUM_QUAD_1D][NUM_QUAD_1D][NUM_QUAD_1D+p_padCubNq];

  dfloat r_Aq[NUM_QUAD_1D];

  const unsigned int t = threadIdx.x;
  const int blk = threadIdx.y;
  
  const int element = blockIdx.x*p_Nblock + blk;
  
  const unsigned int a = t%NUM_DOFS_1D;
  const unsigned int b = t/NUM_DOFS_1D;

#if READ_TO_REGISTER==1
  if(element < numElements){
    for(int c=0;c<NUM_DOFS_1D;++c){
      
      int id = ijklN(a,b,c,element,NUM_DOFS_1D);
      
      r_Aq[c] = solIn[id];
    }
  }
#else
  int n = t + blk*NUM_QUAD_2D;
  
  while(n<p_Nblock*NUM_DOFS_3D){
    
    int id = n + blockIdx.x*p_Nblock*NUM_DOFS_3D;
    if(id<numElements*NUM_DOFS_3D){
      s_tmp1[0][0][0][n] = solIn[id];
    }
    n+=NUM_QUAD_2D*p_Nblock;
  }
#endif
  
  __syncthreads();

#if READ_TO_REGISTER==0
  if(t<NUM_DOFS_2D)
    for(int c=0;c<NUM_DOFS_1D;++c)
      r_Aq[c] = s_tmp1[blk][c][b][a];

  __syncthreads();
#endif
  
  BK1MonolithicDevice  <NUM_DOFS_1D, NUM_QUAD_1D, p_Nblock>
    (numElements, element, op, DofToQuad, s_tmp1, r_Aq);

#if USE_CONTIGUOUS_OUTPUT==0

  if(element<numElements && t<NUM_DOFS_2D){
#pragma unroll
    for(int c=0;c<NUM_DOFS_1D;++c){
      int id = ijklN(a,b,c,element,NUM_DOFS_1D);
      solOut[id] = r_Aq[c];
    }
  }

#else
  
  int n = t + blk*NUM_QUAD_2D;
  
  while(n<p_Nblock*NUM_DOFS_3D){
    
    int id = n + blockIdx.x*p_Nblock*NUM_DOFS_3D;
    if(id<numElements*NUM_DOFS_3D){
      solOut[id] = s_tmp1[0][0][0][n];
    }
    n+=NUM_QUAD_2D*p_Nblock;
  }
#endif  

  
}


template <int NUM_DOFS_1D, int NUM_QUAD_1D, int p_Nblock >
  __global__ void BK1MonolithicConstantKernel(const int numElements,
					      const dfloat * __restrict__ op,
					      const dfloat * __restrict__ DofToQuad,
					      const dfloat * __restrict__ evenDofToQuad,
					      const dfloat * __restrict__ solIn,
					      dfloat * __restrict__ solOut){
  
  __shared__ dfloat s_tmp1[p_Nblock][NUM_QUAD_1D][NUM_QUAD_1D][NUM_QUAD_1D+p_padCubNq];
  
  dfloat r_Aq[NUM_QUAD_1D];

  const unsigned int t = threadIdx.x;
  const int blk = threadIdx.y;
  
  const int element = blockIdx.x*p_Nblock + blk;
  
  const unsigned int a = t%NUM_DOFS_1D;
  const unsigned int b = t/NUM_DOFS_1D;

#if READ_TO_REGISTER==1
  if(element < numElements){
    for(int c=0;c<NUM_DOFS_1D;++c){
      
      int id = ijklN(a,b,c,element,NUM_DOFS_1D);
      
      r_Aq[c] = solIn[id];
    }
  }
#else
  int n = t + blk*NUM_QUAD_2D;
  
  while(n<p_Nblock*NUM_DOFS_3D){
    
    int id = n + blockIdx.x*p_Nblock*NUM_DOFS_3D;
    if(id<numElements*NUM_DOFS_3D){
      s_tmp1[0][0][0][n] = solIn[id];
    }
    n+=NUM_QUAD_2D*p_Nblock;
  }
#endif
  
  __syncthreads();

#if READ_TO_REGISTER==0
  if(t<NUM_DOFS_2D)
    for(int c=0;c<NUM_DOFS_1D;++c)
      r_Aq[c] = s_tmp1[blk][c][b][a];

  __syncthreads();
#endif
  
  BK1MonolithicDevice  <NUM_DOFS_1D, NUM_QUAD_1D, p_Nblock>
    (numElements, element, op, const_DofToQuad, s_tmp1, r_Aq);

#if USE_CONTIGUOUS_OUTPUT==0

  if(element<numElements && t<NUM_DOFS_2D){
#pragma unroll
    for(int c=0;c<NUM_DOFS_1D;++c){
      int id = ijklN(a,b,c,element,NUM_DOFS_1D);
      solOut[id] = r_Aq[c];
    }
  }

#else
  
  int n = t + blk*NUM_QUAD_2D;
  
  while(n<p_Nblock*NUM_DOFS_3D){
    
    int id = n + blockIdx.x*p_Nblock*NUM_DOFS_3D;
    if(id<numElements*NUM_DOFS_3D){
      solOut[id] = s_tmp1[0][0][0][n];
    }
    n+=NUM_QUAD_2D*p_Nblock;
  }
#endif  

  
}

template <int NUM_DOFS_1D, int NUM_QUAD_1D>
  __forceinline__ __device__ 
  void BK1CubeDevice(const int numElements,
		     const int element,
		     const dfloat * __restrict__ op,
		     const dfloat * __restrict__ DofToQuad,
		     dfloat s_p[NUM_QUAD_1D][NUM_QUAD_1D][NUM_QUAD_1D+p_padCubNq]){
  
  const int a = threadIdx.x;
  const int b = threadIdx.y;
  const int c = threadIdx.z;

  // assume s_p preloaded
  
  __syncthreads();
  
  // transform in 'a'
  dfloat res = 0;
  if(b<NUM_DOFS_1D && c<NUM_DOFS_1D){
#pragma unroll 
    for(int n=0;n<NUM_DOFS_1D;++n){
      int an = ijN(n,a,NUM_DOFS_1D);		
      res  += DofToQuad[an]*s_p[c][b][n];
    }
  }
  
  __syncthreads();
  
  if(b<NUM_DOFS_1D && c<NUM_DOFS_1D){
    s_p[c][b][a]  = res;
  }
  
  __syncthreads();
  
  // transform in 'b'
  res = 0;
  
  if(c<NUM_DOFS_1D){
#pragma unroll 
    for(int n=0;n<NUM_DOFS_1D;++n){
      int bn = ijN(n,b,NUM_DOFS_1D);		
      res  += DofToQuad[bn]*s_p[c][n][a];
    }
  }
  
  __syncthreads();

  if(c<NUM_DOFS_1D){
    s_p[c][b][a]  = res;
  }
  
  __syncthreads();

  int gid = ijklN(a,b,c,element, NUM_QUAD_1D);
  dfloat WJ = op[gid];
  
  // transform in 'c'
  res = 0;

#pragma unroll 
  for(int n=0;n<NUM_DOFS_1D;++n){
    int cn = ijN(n,c,NUM_DOFS_1D);		
    res  += DofToQuad[cn]*s_p[n][b][a];
  }

  __syncthreads();
  
  s_p[c][b][a] = WJ*res;
  
  __syncthreads();
  
  // test in 'c'
  res = 0;
  if(c<NUM_DOFS_1D){
#pragma unroll 
    for(int n=0;n<NUM_QUAD_1D;++n){
      int cn = ijN(c,n,NUM_DOFS_1D);		
      res  += DofToQuad[cn]*s_p[n][b][a];
    }
  }

  __syncthreads();

  if(c<NUM_DOFS_1D){
    s_p[c][b][a] = res;
  }

  __syncthreads();


  // transform in 'b'
  res = 0;
  if(b<NUM_DOFS_1D && c<NUM_DOFS_1D){
#pragma unroll 
    for(int n=0;n<NUM_QUAD_1D;++n){
      int bn = ijN(b,n,NUM_DOFS_1D);		
      res  += DofToQuad[bn]*s_p[c][n][a];
    }
  }
  
  __syncthreads();
  
  if(b<NUM_DOFS_1D && c<NUM_DOFS_1D){
    s_p[c][b][a]  = res;
  }

  __syncthreads();
  
  // test in 'a'
  res = 0;
  if(a<NUM_DOFS_1D && b<NUM_DOFS_1D && c<NUM_DOFS_1D){
#pragma unroll 
    for(int n=0;n<NUM_QUAD_1D;++n){
      int an = ijN(a,n,NUM_DOFS_1D);		
      res  += DofToQuad[an]*s_p[c][b][n];
    }
  }

  __syncthreads();
  
  if(a<NUM_DOFS_1D && b<NUM_DOFS_1D && c<NUM_DOFS_1D){
    s_p[c][b][a] = res;
  }

  __syncthreads();
}


template <int NUM_DOFS_1D, int NUM_QUAD_1D>
  __global__ void BK1CubeKernel(const int numElements,
				const dfloat * __restrict__ op,
				const dfloat * __restrict__ DofToQuad,
				const dfloat * __restrict__ solIn,
				dfloat * __restrict__ solOut){
  
  __shared__ dfloat s_p[NUM_QUAD_1D][NUM_QUAD_1D][NUM_QUAD_1D+p_padCubNq];
  __shared__ dfloat s_DofToQuad[NUM_QUAD_1D*NUM_DOFS_1D];
  
  const int element = blockIdx.x;
  
  const int i = threadIdx.x;
  const int j = threadIdx.y;
  const int k = threadIdx.z;

  int t = i + j*NUM_QUAD_1D + k*NUM_QUAD_2D;

  int a = t%NUM_DOFS_1D;
  int b = (t/NUM_DOFS_1D)%NUM_DOFS_1D;
  int c = (t/NUM_DOFS_2D);

  int id = ijklN(a,b,c,element,NUM_DOFS_1D);
    
  if(t<NUM_DOFS_3D){
    s_p[c][b][a] = solIn[t + element*NUM_DOFS_3D];
  }

  if(t<NUM_DOFS_1D*NUM_QUAD_1D){
    s_DofToQuad[t] = DofToQuad[t];
  }
  
  BK1CubeDevice  <NUM_DOFS_1D, NUM_QUAD_1D>
    (numElements, element, op, s_DofToQuad, s_p);
  
  if(t<NUM_DOFS_3D){
    solOut[t + element*NUM_DOFS_3D] = s_p[c][b][a] ;
  }
}




void buildInterpMatrices(int NUM_DOFS_1D, int NUM_QUAD_1D,
			 dfloat *h_DofToQuad,     dfloat *h_oddDofToQuad, dfloat *h_evenDofToQuad,
			 dfloat **c_oddDofToQuad, dfloat **c_evenDofToQuad){

  dfloat *X = (dfloat*) calloc(NUM_DOFS_1D*NUM_DOFS_1D, sizeof(dfloat));
  dfloat *invX = (dfloat*) calloc(NUM_DOFS_1D*NUM_DOFS_1D, sizeof(dfloat));

  dfloat *cubX = (dfloat*) calloc(NUM_QUAD_1D*NUM_QUAD_1D, sizeof(dfloat));
  dfloat *cubInvX = (dfloat*) calloc(NUM_QUAD_1D*NUM_QUAD_1D, sizeof(dfloat));

  for(int n=0;n<NUM_QUAD_1D;++n){
    cubX[n*NUM_QUAD_1D + n] = 1;
    cubInvX[n*NUM_QUAD_1D + n] = 0.5;

    if(n<NUM_QUAD_1D/2){
      cubX[n*NUM_QUAD_1D + NUM_QUAD_1D-1-n] = -1;
      cubInvX[n*NUM_QUAD_1D + NUM_QUAD_1D-1-n] = +0.5;
    }
    
    if(n>=(NUM_QUAD_1D/2)){
      cubX[n*NUM_QUAD_1D + NUM_QUAD_1D-1-n] = +1;
      cubInvX[n*NUM_QUAD_1D + NUM_QUAD_1D-1-n] = -0.5;
    }
  }

  for(int n=0;n<NUM_DOFS_1D;++n){
    X[n*NUM_DOFS_1D + n] = 1;
    invX[n*NUM_DOFS_1D + n] = 0.5;

    if(n<NUM_DOFS_1D/2){
      X[n*NUM_DOFS_1D + NUM_DOFS_1D-1-n] = 1;
      invX[n*NUM_DOFS_1D + NUM_DOFS_1D-1-n] = -0.5;
    }
    
    if(n>=NUM_DOFS_1D/2){
      X[n*NUM_DOFS_1D + NUM_DOFS_1D-1-n] = -1;
      invX[n*NUM_DOFS_1D + NUM_DOFS_1D-1-n] = 0.5;
    }
  }

  if(NUM_DOFS_1D%2) X[(NUM_DOFS_1D)*(NUM_DOFS_1D)/2] = 1;
  if(NUM_DOFS_1D%2) invX[(NUM_DOFS_1D)*(NUM_DOFS_1D)/2] = 1;
  
  if(NUM_QUAD_1D%2) cubX[(NUM_QUAD_1D)*(NUM_QUAD_1D)/2] = 1;
  if(NUM_QUAD_1D%2) cubInvX[(NUM_QUAD_1D)*(NUM_QUAD_1D)/2] = 1;

  matrixPrint(NUM_DOFS_1D, NUM_DOFS_1D, X, "X");
  matrixPrint(NUM_QUAD_1D, NUM_QUAD_1D, cubX, "cubX");

  
  matrixPrint(NUM_DOFS_1D, NUM_DOFS_1D, invX, "invX");
  matrixPrint(NUM_QUAD_1D, NUM_QUAD_1D, cubInvX, "cubInvX");

  
  dfloat *IinvX = (dfloat*) calloc(NUM_DOFS_1D*NUM_QUAD_1D, sizeof(dfloat));
  dfloat *cubInvXIinvX = (dfloat*) calloc(NUM_DOFS_1D*NUM_QUAD_1D, sizeof(dfloat));

  // post multiply by invX
  for(int i=0;i<NUM_QUAD_1D;++i){
    for(int a=0;a<NUM_DOFS_1D;++a){
      dfloat res = 0;
      for(int n=0;n<NUM_DOFS_1D;++n){
	res += h_DofToQuad[i*NUM_DOFS_1D+n]*invX[n*NUM_DOFS_1D+a];
      }
      IinvX[i*NUM_DOFS_1D+a] = res;
    }
  }

  matrixPrint(NUM_QUAD_1D, NUM_DOFS_1D, IinvX, "IinvX");

  // pre multiply by invX
  for(int i=0;i<NUM_QUAD_1D;++i){
    for(int a=0;a<NUM_DOFS_1D;++a){
      dfloat res = 0;
      for(int n=0;n<NUM_QUAD_1D;++n){
	res += cubInvX[i*NUM_QUAD_1D+n]*IinvX[n*NUM_DOFS_1D + a];
      }
      cubInvXIinvX[i*NUM_DOFS_1D+a] = res;
    }
  }

  matrixPrint(NUM_QUAD_1D, NUM_DOFS_1D, cubInvXIinvX, "cubInvXIinvX");
  
  
  for(int i=0;i<HALF_QUAD_1D;++i){
    for(int a=0;a<HALF_DOFS_1D;++a){

      h_oddDofToQuad[i*HALF_DOFS_1D+a] = cubInvXIinvX[i*NUM_DOFS_1D+a];

      h_evenDofToQuad[i*HALF_DOFS_1D+a] = cubInvXIinvX[(NUM_QUAD_1D-1-i)*NUM_DOFS_1D + NUM_DOFS_1D-1-a];
      
    }
  }

  if((NUM_QUAD_1D%2)) // zero duplicate
    h_evenDofToQuad[HALF_QUAD_1D*HALF_DOFS_1D-1] = 0;

  matrixPrint(HALF_QUAD_1D, HALF_DOFS_1D, h_oddDofToQuad, "h_oddDofToQuad");
  matrixPrint(HALF_QUAD_1D, HALF_DOFS_1D, h_evenDofToQuad, "h_evenDofToQuad");
  
  int NoddDofToQuad = HALF_QUAD_1D*HALF_DOFS_1D;
  int NevenDofToQuad = HALF_QUAD_1D*HALF_DOFS_1D;
  
  hipMalloc(c_oddDofToQuad, NoddDofToQuad*sizeof(dfloat));
  hipMalloc(c_evenDofToQuad, NevenDofToQuad*sizeof(dfloat));
  
  hipMemcpy(*c_oddDofToQuad,  h_oddDofToQuad,  NoddDofToQuad*sizeof(dfloat),  hipMemcpyHostToDevice);
  hipMemcpy(*c_evenDofToQuad, h_evenDofToQuad, NoddDofToQuad*sizeof(dfloat), hipMemcpyHostToDevice);
  
  hipMemcpyToSymbol(CUDA_SYMBOL(const_oddDofToQuad),  h_oddDofToQuad,  NoddDofToQuad*sizeof(dfloat));
  hipMemcpyToSymbol(CUDA_SYMBOL(const_evenDofToQuad), h_evenDofToQuad, NoddDofToQuad*sizeof(dfloat));
  hipMemcpyToSymbol(CUDA_SYMBOL(const_DofToQuad),     h_DofToQuad, NUM_QUAD_1D*NUM_DOFS_1D*sizeof(dfloat));
}


void runBK1Kernel(hipStream_t stream, int Nq, int cubNq, int numElements,
				 dfloat *c_op,
				 dfloat *c_DofToQuad, dfloat *c_oddDofToQuad, dfloat *c_evenDofToQuad,
				 dfloat *c_solIn, dfloat *c_solOut, int mode){
  
#define BK1Kernel(Nq,cubNq,Nblock)					\
  {									\
    dim3 G((numElements+Nblock-1)/Nblock, 1, 1);			\
    dim3 B(cubNq*cubNq, Nblock, 1);					\
    									\
    if(mode==1)								\
      BK1RegisterKernel<Nq,cubNq,Nblock> <<< G, B, 0, stream >>>( numElements, c_op, c_oddDofToQuad, c_evenDofToQuad, c_solIn, c_solOut); \
    else if(mode==2)							\
      BK1ConstantKernel<Nq,cubNq,Nblock> <<< G, B, 0, stream >>>( numElements, c_op, c_oddDofToQuad, c_evenDofToQuad, c_solIn, c_solOut); \
    else if(mode==3)							\
      BK1SharedKernel<Nq,cubNq,Nblock> <<< G, B, 0, stream >>>( numElements, c_op, c_oddDofToQuad, c_evenDofToQuad, c_solIn, c_solOut); \
    else if(mode==4)							\
      BK1GlobalKernel<Nq,cubNq,Nblock> <<< G, B, 0, stream >>>( numElements, c_op, c_oddDofToQuad, c_evenDofToQuad, c_solIn, c_solOut); \
    else if(mode==5)							\
      BK1MonolithicGlobalKernel<Nq,cubNq,Nblock> <<< G, B, 0, stream >>>( numElements, c_op, c_DofToQuad, c_evenDofToQuad, c_solIn, c_solOut); \
    else if(mode==6)							\
      BK1MonolithicConstantKernel<Nq,cubNq,Nblock> <<< G, B, 0, stream >>>( numElements, c_op, c_DofToQuad, c_evenDofToQuad, c_solIn, c_solOut); \
    else if(mode==7){							\
      dim3 G3(numElements, 1, 1);					\
      dim3 B3(cubNq,cubNq,cubNq);					\
      BK1CubeKernel<Nq,cubNq> <<< G3, B3, 0, stream >>>( numElements, c_op, c_DofToQuad, c_solIn, c_solOut); \
    }									\
  }
  
#define ERR printf("BK1Register with Nq=%d, cubNq=%d not available", Nq, cubNq); exit(-1)

  if(Nq==2){
    switch(cubNq){
    case 2: BK1Kernel(2,2,16); break;
    case 3: BK1Kernel(2,3, 7); break;
    case 4: BK1Kernel(2,4, 4); break;
    case 5: BK1Kernel(2,5, 5); break;
    case 6: BK1Kernel(2,6, 3); break;
    default: ERR;
    }
    return;
  }

  if(Nq==3){
    switch(cubNq){
    case 3: BK1Kernel(3,3,7); break;
    case 4: BK1Kernel(3,4,16); break;
    case 5: BK1Kernel(3,5,5); break;
    case 6: BK1Kernel(3,6,3); break;
    case 7: BK1Kernel(3,7,2); break;
    default: ERR;
    }
    return;
  }

  if(Nq==4){
    switch(cubNq){
    case 4: BK1Kernel(4,4,4); break;
    case 5: BK1Kernel(4,5,5); break;
    case 6: BK1Kernel(4,6,3); break;
    case 7: BK1Kernel(4,7,2); break;
    case 8: BK1Kernel(4,8,1); break;
    default: ERR;
    }
    return;
  }

  if(Nq==5){
    switch(cubNq){
    case 5: BK1Kernel(5,5,5); break;
    case 6: BK1Kernel(5,6,3); break;
    case 7: BK1Kernel(5,7,2); break;
    case 8: BK1Kernel(5,8,1); break;
    case 9: BK1Kernel(5,9,2); break;
    default: ERR;
    }
    return;
  }

  if(Nq==6){
    switch(cubNq){
    case 6:  BK1Kernel(6, 6, 3); break; // Nb=3 best so far
    case 7:  BK1Kernel(6, 7, 2); break;
    case 8:  BK1Kernel(6, 8, 1); break;
    case 9:  BK1Kernel(6, 9, 2); break;
    case 10: BK1Kernel(6,10, 1); break;
    default: ERR;
    }
    return;
  }

  if(Nq==7){
    switch(cubNq){
    case 7:  BK1Kernel(7, 7,2); break;
    case 8:  BK1Kernel(7, 8,1); break;
    case 9:  BK1Kernel(7, 9,2); break;
    case 10: BK1Kernel(7,10,1); break;
    case 11: BK1Kernel(7,11,1); break;

    default: ERR;
    }
    return;
  }

  if(Nq==8){
    switch(cubNq){
    case 8:  BK1Kernel(8, 8,1); break;
    case 9:  BK1Kernel(8, 9,2); break;
    case 10: BK1Kernel(8,10,1); break;
    case 11: BK1Kernel(8,11,1); break;
    case 12: BK1Kernel(8,12,1); break;
    default: ERR;
    }
    return;
  }

  if(Nq==9){
    switch(cubNq){
    case 9:  BK1Kernel(9, 9,1); break;
    case 10: BK1Kernel(9,10,1); break;
    case 11: BK1Kernel(9,11,1); break;
    case 12: BK1Kernel(9,12,1); break;
    case 13: BK1Kernel(9,13,1); break;

    default: ERR;
    }
    return;
  }

  if(Nq==10){
    switch(cubNq){
    case 10: BK1Kernel(10,10,1); break;
    case 11: BK1Kernel(10,11,1); break;
    case 12: BK1Kernel(10,12,1); break;
    case 13: BK1Kernel(10,13,1); break;
    case 14: BK1Kernel(10,14,1); break;
    default: ERR;
    }
    return;
  }

  if(Nq==11){
    switch(cubNq){
    case 11: BK1Kernel(11,11,1); break;
    case 12: BK1Kernel(11,12,1); break;
    case 13: BK1Kernel(11,13,1); break;
    case 14: BK1Kernel(11,14,1); break;
    case 15: BK1Kernel(11,15,1); break;

    default: ERR;
    }
    return;
  }
  
  if(Nq==12){
    switch(cubNq){
    case 12: BK1Kernel(12,12,1); break;
    case 13: BK1Kernel(12,13,1); break;
    case 14: BK1Kernel(12,14,1); break;
    case 15: BK1Kernel(12,15,1); break;
      //    case 16: BK1Kernel(12,16,1); break;
    default: ERR;
    }
    return;
  }
  
  ERR;
}


dfloat nothingTest(hipStream_t stream, int Ntests){

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);	

  hipDeviceSynchronize();
  
  float nothingElapsed = 0;
  {
    
    // time kernel that does nothing
    
#if USE_GRAPH==1
    // cuda stream capture sequence for nothingKernel
    hipGraph_t nothingGraph;
    
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    
    for(int test=0;test<Ntests;++test){
      nothingKernel <<< 1, 1, 0, stream >>> ();
    }
    
    hipStreamEndCapture(stream, &nothingGraph);
    
    // time graph sequence for nothing
    hipGraphExec_t nothingInstance;
    hipGraphInstantiate(&nothingInstance, nothingGraph, NULL, NULL, 0);
    
    hipEventRecord(start, stream);
    
    hipGraphLaunch(nothingInstance, stream);
    
    hipEventRecord(end, stream);
#else
    
    hipEventRecord(start, stream);
    
    for(int test=0;test<Ntests;++test)
      nothingKernel <<< 1, 1, 0, stream >>> ();
    
    
    hipEventRecord(end, stream);
    
#endif
    
    hipDeviceSynchronize();
    
    hipEventElapsedTime(&nothingElapsed, start, end);
    nothingElapsed /= 1000.;
    nothingElapsed /= (double) Ntests;
    
  }

  return nothingElapsed;
}


double bandwidthTest(hipStream_t stream, int Ntests, size_t bwNtotal){

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);	
  
  dfloat *h_bwTest1, *c_bwTest1;
  dfloat *h_bwTest2, *c_bwTest2;
  
  randAlloc(bwNtotal/2, &h_bwTest1, &c_bwTest1);
  randAlloc(bwNtotal/2, &h_bwTest2, &c_bwTest2);
  
  hipDeviceSynchronize();
  hipEventRecord(start, stream);
  
  for(int test=0;test<Ntests/2;++test){
    hipMemcpy(c_bwTest2, c_bwTest1, (bwNtotal/2)*sizeof(dfloat), hipMemcpyDeviceToDevice);
    hipMemcpy(c_bwTest1, c_bwTest2, (bwNtotal/2)*sizeof(dfloat), hipMemcpyDeviceToDevice);
  }
  
  hipEventRecord(end, stream);
  hipEventSynchronize(end);
  hipDeviceSynchronize();

  float elapsed;
  hipEventElapsedTime(&elapsed, start, end);
  elapsed /= 1000.; // convert to s
  elapsed /= (double) Ntests;
  
  double estimatedActualDeviceBandwidth = (bwNtotal*sizeof(dfloat)/elapsed)/1.e9;
  
  hipFree(c_bwTest1);
  hipFree(c_bwTest2);
  
  free(h_bwTest1);
  free(h_bwTest2);
  
  hipEventDestroy(start);
  hipEventDestroy(end);	
  
  return estimatedActualDeviceBandwidth;
}


int main(int argc, char **argv){

  hipSetDevice(0);
  
  hipStream_t stream;
  hipStreamCreate(&stream);
  
  if(argc!=5){
    printf("Usage: ./BK1VT Nq cubNq numElements mode \n");
    exit(-1);
  }

  // read number of elements
  int        Nq = atoi(argv[1]);
  int     cubNq = atoi(argv[2]);
  int numElements = atoi(argv[3]);
  int        mode = atoi(argv[4]);

  if(mode==0 || mode>7) {
    printf("Exiting: mode %d not supported\n", mode);
  }
  
  printf("Running: NUM_DOFS_1D=%d, NUM_QUAD_1D=%d, numElements=%d, mode=%d\n", Nq, cubNq, numElements, mode);

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);	

  int Ntests = 50;
  
  // do nothing kernel test
  dfloat nothingElapsed = nothingTest(stream, Ntests);
  nothingElapsed = nothingTest(stream, Ntests);

  int   Np = Nq*Nq*Nq;
  int   cubNp = cubNq*cubNq*cubNq;

  int halfNq = ((Nq+1)/2);
  int halfCubNq = ((cubNq+1)/2);

  int    Ntotal = numElements*Np;
  int cubNtotal = numElements*cubNp;

  // bandwidth test
  // total number of bytes

  double estimatedActualDeviceBandwidth = bandwidthTest(stream, Ntests, (Ntotal*2+cubNtotal)*sizeof(dfloat));
  
  dfloat *h_op,      *c_op;
  dfloat *h_solOut,       *c_solOut;
  dfloat *h_solIn,        *c_solIn;
  dfloat *h_DofToQuad,    *c_DofToQuad;
  dfloat *h_oddDofToQuad, *c_oddDofToQuad;
  dfloat *h_evenDofToQuad, *c_evenDofToQuad;


  // float fields
  randAlloc(cubNtotal*p_Nvgeo, &h_op, &c_op);

  for(int e=0;e<numElements;++e){
    for(int n=0;n<cubNp;++n){
      h_op[e*cubNp+n] = drand48();
    }
  }
  
  hipMemcpy(c_op, h_op, p_Nvgeo*numElements*cubNp*sizeof(dfloat), hipMemcpyHostToDevice);
  
  randAlloc(Ntotal, &h_solIn, &c_solIn);
  randAlloc(Ntotal, &h_solOut, &c_solOut);
  
  randAlloc(Nq*cubNq, &h_DofToQuad, &c_DofToQuad);
  randAlloc(halfNq*halfCubNq, &h_oddDofToQuad, &c_oddDofToQuad);
  randAlloc(halfNq*halfCubNq, &h_evenDofToQuad, &c_evenDofToQuad);

  // build interpolation matrix
  dfloat *r, *w, *cubr, *cubw;
  meshJacobiGL(0,0,Nq-1, &r, &w);
  meshJacobiGQ(0,0,cubNq-1, &cubr, &cubw);
  meshInterpolationMatrix1D(Nq-1, Nq, r, cubNq, cubr, &h_DofToQuad);
  hipMemcpy(c_DofToQuad, h_DofToQuad, cubNq*Nq*sizeof(dfloat), hipMemcpyHostToDevice);
  
  matrixPrint(cubNq, Nq, h_DofToQuad, "DofToQuad");

  // create Odd-even packed storage for I and transpose(I) and push to constant memory
  buildInterpMatrices (Nq,cubNq, h_DofToQuad, h_oddDofToQuad, h_evenDofToQuad,
		       &c_oddDofToQuad, &c_evenDofToQuad);


  // KERNEL GRID
  float elapsed;
  
  // warm up call
  runBK1Kernel (stream, Nq, cubNq, numElements, c_op, c_DofToQuad, c_oddDofToQuad, c_evenDofToQuad, c_solIn, c_solOut, mode);

  hipDeviceSynchronize();

  {
    hipEventRecord(start, stream);
    
    for(int test=0;test<Ntests;++test)
      runBK1Kernel (stream, Nq, cubNq, numElements, c_op, c_DofToQuad, c_oddDofToQuad, c_evenDofToQuad, c_solIn, c_solOut, mode);

    hipEventRecord(end, stream);

    hipDeviceSynchronize();
    
    hipEventElapsedTime(&elapsed, start, end);
    elapsed /= 1000.;
    elapsed /= (double) Ntests;

    // estimate bandwidth (assuming all data moved to/from device memory)
    int bytesMoved = (2*Np+cubNp)*sizeof(dfloat); // x, Mx, opa   
    double bw = (bytesMoved*numElements/elapsed)/1.e9;

    double estFlops =
      numElements*(( Nq*Nq*(halfNq*2 + halfCubNq*(halfNq*4 +2)) +
		     Nq*cubNq*(halfNq*2 + halfCubNq*(halfNq*4 + 2)) + 
		     cubNq*cubNq*(halfNq*2 + halfCubNq*(halfNq*4+6) + halfNq*(halfCubNq*4+2)) + 
		     Nq*cubNq*(halfNq*2+halfNq*(halfCubNq*4 + 2)) +
		     Nq*Nq*(halfCubNq*2 + halfNq*(halfCubNq*4 + 2)))/elapsed)/1.e9;

    double effectiveFlops =
      numElements*(2*( Nq*Nq*Nq*cubNq*2 + Nq*Nq*cubNq*cubNq*2 + Nq*cubNq*cubNq*cubNq*2)/elapsed)/1.e9;
    
    printf("%2d %2d %8d %8d %e %e %e %e %e %e %e %d %%%% [BK1: NUM_DOFS_1D, NUM_QUAD_1D, numElements, Ndofs,"
	   " elapsed, dofsPerSecond, nothingElapsed, BW in GB/s, estimated peak Device BW, est. GFLOPS/s, oddeven GFLOPS/s, mode]\n",
	   Nq, cubNq, numElements, Np*numElements, elapsed, numElements*(Np/elapsed), nothingElapsed, bw, estimatedActualDeviceBandwidth, estFlops, effectiveFlops, mode);
  }

  // check output is correct
  meshReferenceBK1(Nq, cubNq, numElements, h_op, h_DofToQuad, h_solIn, h_solOut);

  // copy device version to host old q
  dfloat *fromDevice = (dfloat*) calloc(numElements*Np, sizeof(dfloat));
  hipMemcpy(fromDevice, c_solOut, numElements*Np*sizeof(dfloat), hipMemcpyDeviceToHost);

  dfloat maxDiff = 0;
  
  for(int e=0;e<numElements;++e){
    for(int n=0;n<Np;++n){
      int id = e*Np + n;
      dfloat diff = fabs(h_solOut[id]-fromDevice[id]);
      maxDiff = (diff>maxDiff) ? diff:maxDiff;
    }
  }

  printf("NUM_DOFS_1D=%02d, NUM_QUAD_1D=%02d || Mq_{host} - Mq_{device} ||_linf = %lg\n", Nq, cubNq, maxDiff);

  hipEventDestroy(start);
  hipEventDestroy(end);	
  
  return 0;

}
